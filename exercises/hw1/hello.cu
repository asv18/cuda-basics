
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void hello() {
  printf("Hello from block: %u, thread: %u\n", blockIdx.x, threadIdx.x);
}

int main() {
  hello<<<2, 2>>>();
  hipError_t err = hipDeviceSynchronize();
  if (err != hipSuccess) {
      printf("CUDA error: %s\n", hipGetErrorString(err));
  }

  return 0;
}

